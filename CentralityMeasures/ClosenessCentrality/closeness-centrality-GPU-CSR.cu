#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void cc_kernel(int n, int *R, int *C, double *cc, int *distance);

int main(int argc, char const *argv[]) {
    
    FILE *R = fopen("data/row_offsets.dat", "r");
    FILE *C = fopen("data/column_indices.dat", "r");

    int n, r_size, c_size;
    int *h_r, *h_c;
    int *d_r, *d_c, *d_dist;
    double *h_cc, *d_cc;

    /* Input: numero di nodi e archi del grafo */
    printf("Inserire numero di nodi: ");
    scanf("%d", &n);
    r_size = n + 1;

    printf("Inserire numero di archi: ");
    scanf("%d", &c_size);

    /* allocazione strutture dati host */
    h_r = (int*)malloc(r_size * sizeof(int));
    h_c = (int*)malloc(c_size * sizeof(int));
    h_cc = (double*)malloc(n * sizeof(double));

    /* allocazione strutture dati device */
    hipMalloc((void **) &d_r, r_size * sizeof(int));
    hipMalloc((void **) &d_c, c_size * sizeof(int));
    hipMalloc((void **) &d_dist, n * sizeof(int));
    hipMalloc((void **) &d_cc, n * sizeof(double));

    /* Leggo da file il columns indices ed il row offsets array */
    for (int i = 0; i < r_size; i++) {
        fscanf(R, "%d\n", &h_r[i]);
    }
    for (int i = 0; i < c_size; i++) {
        fscanf(C, "%d\n", &h_c[i]);
    }

    /* copia gli array row_offests e column_indices sul device */
    hipMemcpy(d_r, h_r, r_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, c_size * sizeof(int), hipMemcpyHostToDevice);

    /* configurazione del Kernel */
    dim3 blockDim(64);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x);
    
    cc_kernel<<<gridDim, blockDim>>>(n, d_r, d_c, d_cc, d_dist);

    hipMemcpy(h_cc, d_cc, n * sizeof(double), hipMemcpyDeviceToHost);

    printf("Closeness Centrality:\n");
    for (int i = 0; i < n; i++) {
        printf("Score %d: %f\n", i+1, h_cc[i]);
    }

    /* free della memoria */
    free(h_r);
    free(h_c);
    free(h_cc);
    hipFree(d_r);
    hipFree(d_c);
    hipFree(d_cc);
    hipFree(d_dist);
    
    return 0;
}

__global__ void cc_kernel(int n, int *R, int *C, double *cc, int *distance) {
    int idx = threadIdx.x;

    if (idx >= n) return;

    /* variabili shared */
    __shared__ int s;
    __shared__ int current_depth;
    __shared__ bool done;

    if (idx == 0) {
        s = -1;
    }
    __syncthreads();

    /* iteriamo per ogni nodo radice s */
    while (s < n - 1) {
        /* solo il thread 0 inizializza le variabili shared */
        if (idx == 0) {
            s++;
            done = false;
            current_depth = -1;
        }
        __syncthreads();

        /* inizializziamo le distanze */
        for (int v = idx; v < n; v += blockDim.x) {
            if (v == s) distance[v] = 0;
            else distance[v] = INT_MAX;
        }
        __syncthreads();

        /*  
            eseguo una BFS per calcolare gli shortest 
            path e la distanza da s a tutti gli altri nodi 
        */
        while (!done) {
            if (idx == 0) {
                current_depth++;
            }
            done = true;
            __syncthreads();

            for (int v = idx; v < n; v += blockDim.x) {
                if (distance[v] == current_depth) {
                    for (int r = R[v]; r < R[v+1]; r++) {
                        int w = C[r];
                        if (distance[w] == INT_MAX) {
                            distance[w] = distance[v] + 1;
                            done = false;
                        }
                    }
                }
            }
            __syncthreads();
        }
        
        if (idx == 0) {
            int dist_sum = 0;
            for (int i = 0; i < n; i++) {
                dist_sum += distance[i];
            }
            cc[s] = (double) n / dist_sum;
        }
        __syncthreads();
    }
}

